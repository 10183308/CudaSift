#include "hip/hip_runtime.h"
//********************************************************//
// CUDA SIFT extractor by Mårten Björkman aka Celebrandil //
//********************************************************//  

#include <cstdio>
#include <cstring>
#include <cmath>
#include <iostream>
#include <cudautils.h>

#include "cudaImage.h"
#include "cudaSift.h"
#include "cudaSiftD.h"
#include "cudaSiftH.h"

#include "cudaSiftD.cu"

void InitCuda()
{
  deviceInit(0);  
}

void ExtractSift(SiftData &siftData, CudaImage &img, int numOctaves, double initBlur, float thresh, float lowestScale, float subsampling) 
{
  TimerGPU timer(0);
  int totPts = 0;
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_PointCounter), &totPts, sizeof(int)));
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_MaxNumPoints), &siftData.maxPts, sizeof(int)));

  const int nb = NUM_SCALES + 3;
  const int nd = NUM_SCALES + 3;
  int w = img.width;
  int h = img.height;
  int p = iAlignUp(w, 128);
  int size = (nb+nd)*h*p;
  int sizeTmp = size;
  for (int i=0;i<numOctaves;i++) {
    w /= 2;
    h /= 2;
    int p = iAlignUp(w, 128);
    size += h*p;
  }
  float *memoryTmp = NULL;
  size_t pitch;
  safeCall(hipMallocPitch((void **)&memoryTmp, &pitch, (size_t)4096, (size+4095)/4096*sizeof(float)));
  float *memorySub = memoryTmp + sizeTmp;

  ExtractSiftLoop(siftData, img, numOctaves, initBlur, thresh, lowestScale, subsampling, memoryTmp, memorySub);
  safeCall(hipMemcpyFromSymbol(&siftData.numPts, HIP_SYMBOL(d_PointCounter), sizeof(int)));
  siftData.numPts = (siftData.numPts<siftData.maxPts ? siftData.numPts : siftData.maxPts);
  safeCall(hipFree(memoryTmp));
#ifdef MANAGEDMEM
  safeCall(hipDeviceSynchronize());
#else
  if (siftData.h_data)
    safeCall(hipMemcpy(siftData.h_data, siftData.d_data, sizeof(SiftPoint)*siftData.numPts, hipMemcpyDeviceToHost));
#endif
  double totTime = timer.read();
#ifndef VERBOSE
  printf("Total time incl memory =      %.2f ms\n", totTime);
#endif
}

void ExtractSiftLoop(SiftData &siftData, CudaImage &img, int numOctaves, double initBlur, float thresh, float lowestScale, float subsampling, float *memoryTmp, float *memorySub) 
{
  TimerGPU timer(0);
  int w = img.width;
  int h = img.height;
  if (numOctaves>1) {
    CudaImage subImg;
    int p = iAlignUp(w/2, 128);
    subImg.Allocate(w/2, h/2, p, false, memorySub); 
    ScaleDown(subImg, img, 0.5f);
    float totInitBlur = (float)sqrt(initBlur*initBlur + 0.5f*0.5f) / 2.0f;
    ExtractSiftLoop(siftData, subImg, numOctaves-1, totInitBlur, thresh, lowestScale, subsampling*2.0f, memoryTmp, memorySub + (h/2)*p);
  }
  if (lowestScale<subsampling*2.0f) 
    ExtractSiftOctave(siftData, img, initBlur, thresh, lowestScale, subsampling, memoryTmp);
  double totTime = timer.read();
#ifdef VERBOSE
  printf("ExtractSift time total =      %.2f ms\n\n", totTime);
#endif
}

void ExtractSiftOctave(SiftData &siftData, CudaImage &img, double initBlur, float thresh, float lowestScale, float subsampling, float *memoryTmp)
{
  const int nb = NUM_SCALES + 3;
  const int nd = NUM_SCALES + 3;
  const double baseBlur = pow(2.0, -1.0/NUM_SCALES);
  int w = img.width; 
  int h = img.height;
  CudaImage blurImg[nb];
  CudaImage diffImg[nd];

  TimerGPU timer0;
  int p = iAlignUp(w, 128);
  for (int i=0;i<nb;i++) 
    blurImg[i].Allocate(w, h, p, false, memoryTmp + i*p*h); 
  for (int i=0;i<nb-1;i++) 
    diffImg[i].Allocate(w, h, p, false, memoryTmp + (nb+i)*p*h); 
  tex.addressMode[0] = hipAddressModeClamp;
  tex.addressMode[1] = hipAddressModeClamp;
  tex.filterMode = hipFilterModeLinear; 
  tex.normalized = false;
  size_t offset = 0;
  safeCall(hipBindTexture2D(&offset, tex, img.d_data, tex.channelDesc, img.width, img.height, img.pitch*sizeof(float)));

  TimerGPU timer1;
  float diffScale = pow(2.0f, 1.0f/NUM_SCALES);
  LaplaceMulti(diffImg, img, baseBlur, diffScale, initBlur);
  int fstPts = 0;
  safeCall(hipMemcpyFromSymbol(&fstPts, HIP_SYMBOL(d_PointCounter), sizeof(int)));
  double sigma = baseBlur*diffScale;
  FindPointsMulti(diffImg, siftData, thresh, 10.0f, sigma, 1.0f/NUM_SCALES, lowestScale/subsampling);
  double gpuTimeDoG = timer1.read();
  TimerGPU timer4;
  int totPts = 0;
  safeCall(hipMemcpyFromSymbol(&totPts, HIP_SYMBOL(d_PointCounter), sizeof(int)));
  totPts = (totPts<siftData.maxPts ? totPts : siftData.maxPts);
  if (totPts>fstPts) {
    ComputeOrientations(img, siftData, fstPts, totPts); 
    safeCall(hipMemcpyFromSymbol(&totPts, HIP_SYMBOL(d_PointCounter), sizeof(int)));
    totPts = (totPts<siftData.maxPts ? totPts : siftData.maxPts);
    ExtractSiftDescriptors(img, siftData, fstPts, totPts, subsampling); 
  }
  safeCall(hipUnbindTexture(tex));
  double gpuTimeSift = timer4.read();

  double totTime = timer0.read();
#ifdef VERBOSE
  printf("GPU time : %.2f ms + %.2f ms + %.2f ms = %.2f ms\n", totTime-gpuTimeDoG-gpuTimeSift, gpuTimeDoG, gpuTimeSift, totTime);
  safeCall(hipMemcpyFromSymbol(&totPts, HIP_SYMBOL(d_PointCounter), sizeof(int)));
  totPts = (totPts<siftData.maxPts ? totPts : siftData.maxPts);
  if (totPts>0) 
    printf("           %.2f ms / DoG,  %.4f ms / Sift,  #Sift = %d\n", gpuTimeDoG/NUM_SCALES, gpuTimeSift/(totPts-fstPts), totPts-fstPts); 
#endif
}

void InitSiftData(SiftData &data, int num, bool host, bool dev)
{
  data.numPts = 0;
  data.maxPts = num;
  int sz = sizeof(SiftPoint)*num;
#ifdef MANAGEDMEM
  safeCall(hipMallocManaged((void **)&data.m_data, sz));
#else
  data.h_data = NULL;
  if (host)
    data.h_data = (SiftPoint *)malloc(sz);
  data.d_data = NULL;
  if (dev)
    safeCall(hipMalloc((void **)&data.d_data, sz));
#endif
}

void FreeSiftData(SiftData &data)
{
#ifdef MANAGEDMEM
  safeCall(hipFree(data.m_data));
#else
  if (data.d_data!=NULL)
    safeCall(hipFree(data.d_data));
  data.d_data = NULL;
  if (data.h_data!=NULL)
    free(data.h_data);
#endif
  data.numPts = 0;
  data.maxPts = 0;
}

void PrintSiftData(SiftData &data)
{
#ifdef MANAGEDMEM
  SiftPoint *h_data = data.m_data;
#else
  SiftPoint *h_data = data.h_data;
  if (data.h_data==NULL) {
    h_data = (SiftPoint *)malloc(sizeof(SiftPoint)*data.maxPts);
    safeCall(hipHostMalloc((void **)&h_data, sizeof(SiftPoint)*data.maxPts));
    safeCall(hipMemcpy(h_data, data.d_data, sizeof(SiftPoint)*data.numPts, hipMemcpyDeviceToHost));
    data.h_data = h_data;
  }
#endif
  for (int i=0;i<data.numPts;i++) {
    printf("xpos         = %.2f\n", h_data[i].xpos);
    printf("ypos         = %.2f\n", h_data[i].ypos);
    printf("scale        = %.2f\n", h_data[i].scale);
    printf("sharpness    = %.2f\n", h_data[i].sharpness);
    printf("edgeness     = %.2f\n", h_data[i].edgeness);
    printf("orientation  = %.2f\n", h_data[i].orientation);
    printf("score        = %.2f\n", h_data[i].score);
    float *siftData = (float*)&h_data[i].data;
    for (int j=0;j<8;j++) {
      if (j==0) 
	printf("data = ");
      else 
	printf("       ");
      for (int k=0;k<16;k++)
	if (siftData[j+8*k]<0.05)
	  printf(" .   ");
	else
	  printf("%.2f ", siftData[j+8*k]);
      printf("\n");
    }
  }
  printf("Number of available points: %d\n", data.numPts);
  printf("Number of allocated points: %d\n", data.maxPts);
}

///////////////////////////////////////////////////////////////////////////////
// Host side master functions
///////////////////////////////////////////////////////////////////////////////

double ScaleDown(CudaImage &res, CudaImage &src, float variance)
{
  if (res.d_data==NULL || src.d_data==NULL) {
    printf("ScaleDown: missing data\n");
    return 0.0;
  }
  float h_Kernel[5];
  float kernelSum = 0.0f;
  for (int j=0;j<5;j++) {
    h_Kernel[j] = (float)expf(-(double)(j-2)*(j-2)/2.0/variance);      
    kernelSum += h_Kernel[j];
  }
  for (int j=0;j<5;j++)
    h_Kernel[j] /= kernelSum;  
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_Kernel), h_Kernel, 5*sizeof(float)));
  dim3 blocks(iDivUp(src.width, SCALEDOWN_W), iDivUp(src.height, SCALEDOWN_H));
  dim3 threads(SCALEDOWN_W + 4);
  ScaleDown<<<blocks, threads>>>(res.d_data, src.d_data, src.width, src.pitch, src.height, res.pitch); 
  checkMsg("ScaleDown() execution failed\n");
  return 0.0;
}

double ComputeOrientations(CudaImage &img, SiftData &siftData, int fstPts, int totPts)
{
  dim3 blocks(totPts - fstPts);
  dim3 threads(128);
#ifdef MANAGEDMEM
  ComputeOrientations<<<blocks, threads>>>(img.d_data, siftData.m_data, fstPts);
#else
  ComputeOrientations<<<blocks, threads>>>(img.d_data, siftData.d_data, fstPts);
#endif
  checkMsg("ComputeOrientations() execution failed\n");
  return 0.0;
}

double ExtractSiftDescriptors(CudaImage &img, SiftData &siftData, int fstPts, int totPts, float subsampling)
{
  dim3 blocks(totPts - fstPts); 
  dim3 threads(16, 8);
#ifdef MANAGEDMEM
  ExtractSiftDescriptors<<<blocks, threads>>>(img.d_data, siftData.m_data, fstPts, subsampling);
#else
  ExtractSiftDescriptors<<<blocks, threads>>>(img.d_data, siftData.d_data, fstPts, subsampling);
#endif
  checkMsg("ExtractSiftDescriptors() execution failed\n");
  return 0.0; 
}

//==================== Multi-scale functions ===================//

double LaplaceMulti(CudaImage *results, CudaImage &origImg, float baseBlur, float diffScale, float initBlur)
{
  float *d_DataA = origImg.d_data;
  float *d_DataB = results[0].d_data;
  if (d_DataA==NULL || d_DataB==NULL) {
    printf("LaplaceMulti: missing data\n");
    return 0.0;
  } 
  float kernel[12*16];
  float scale = baseBlur;
  for (int i=0;i<NUM_SCALES+3;i++) {
    float kernelSum = 0.0f;
    float var = scale*scale - initBlur*initBlur;
    for (int j=-LAPLACE_R;j<=LAPLACE_R;j++) {
      kernel[16*i+j+LAPLACE_R] = (float)expf(-(double)j*j/2.0/var);
      kernelSum += kernel[16*i+j+LAPLACE_R]; 
    }
    for (int j=-LAPLACE_R;j<=LAPLACE_R;j++) 
      kernel[16*i+j+LAPLACE_R] /= kernelSum;  
    scale *= diffScale;
  }
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_Kernel), kernel, 12*16*sizeof(float)));
  int width = results[0].width;
  int pitch = results[0].pitch;
  int height = results[0].height;
  dim3 blocks(iDivUp(width+2*LAPLACE_R, LAPLACE_W), height);
  dim3 threads(LAPLACE_W+2*LAPLACE_R, LAPLACE_S); 
  LaplaceMulti<<<blocks, threads>>>(d_DataB, d_DataA, width, pitch, height);
  checkMsg("ConvRowGPU() execution failed\n");
  return 0.0; 
}

double FindPointsMulti(CudaImage *sources, SiftData &siftData, float thresh, float edgeLimit, float scale, float factor, float lowestScale)
{
  if (sources->d_data==NULL) {
    printf("FindPointsMulti: missing data\n");
    return 0.0;
  }
  int w = sources->width;
  int p = sources->pitch;
  int h = sources->height;
  float threshs[2] = { thresh, -thresh };
  float scales[NUM_SCALES];  
  int nScales = 0;
  float diffScale = pow(2.0f, factor);
  for (int i=0;i<NUM_SCALES;i++) {
    if (scale>=lowestScale) 
      scales[nScales++] = scale;
    scale *= diffScale;
  }
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_Threshold), &threshs, 2*sizeof(float)));
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_EdgeLimit), &edgeLimit, sizeof(float)));
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_Scales), scales, sizeof(float)*NUM_SCALES));
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_Factor), &factor, sizeof(float)));

  if (nScales>0) {
    dim3 blocks(iDivUp(w, MINMAX_W)*nScales, iDivUp(h, MINMAX_H));
    dim3 threads(MINMAX_W + 2); 
#ifdef MANAGEDMEM
    FindPointsMulti<<<blocks, threads>>>(sources->d_data, siftData.m_data, w, p, h, nScales); 
#else
    FindPointsMulti<<<blocks, threads>>>(sources->d_data, siftData.d_data, w, p, h, nScales); 
#endif
    checkMsg("FindPointsMulti() execution failed\n");
  }
  return 0.0;
}

